
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel(void){
	printf("From GPU Hello, world!\n");
}

int main( void ) {
	kernel<<<1,10>>>();
	hipDeviceReset();
	printf("From CPU Hello, world!\n");

	return 0;
}
