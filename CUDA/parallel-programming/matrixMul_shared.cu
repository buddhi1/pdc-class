
#include <hip/hip_runtime.h>
#include<stdio.h>

#define HEIGHT 2<<10
#define WIDTH 2<<10
// Thread block size
#define BLOCK_SIZE 16

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
    int width;
    int height;
    int stride; 
    float* elements;
} Matrix;

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
    A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col) 
{
    Matrix Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                         + BLOCK_SIZE * col];
    return Asub;
}

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // timing parameters ready
    hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;


    // Load A and B to device memory
    Matrix d_A;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);
    Matrix d_B;
    d_B.width = d_B.stride = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
    hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = d_C.stride = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    
	hipEventRecord(start);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	hipEventRecord(stop);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\n\nGPU running time: %f\n\n",milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

// Matrix multiplication kernel called by MatMul()
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes one element of Csub
    // by accumulating results into Cvalue
    float Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

        // Get sub-matrix Asub of A
        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Get sub-matrix Bsub of B
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();
        // Multiply Asub and Bsub together
        for (int e = 0; e < BLOCK_SIZE; ++e)
            Cvalue += As[row][e] * Bs[e][col];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    SetElement(Csub, row, col, Cvalue);
}

int main() {
    Matrix A, B, C;
    int i, j;

    A.elements = (float *)malloc(sizeof(float)*HEIGHT*WIDTH);
    B.elements = (float *)malloc(sizeof(float)*HEIGHT*WIDTH);
    C.elements = (float *)malloc(sizeof(float)*HEIGHT*WIDTH);

    // populate matrix A  and B
    for(i=0; i<HEIGHT ; ++i) {
        for(j=0; j<WIDTH; ++j) {
            A.elements[i*HEIGHT + j] = i;
            B.elements[i*HEIGHT + j] = j;
        }
    }
    A.width = WIDTH;
    B.width = WIDTH;
    A.height = HEIGHT;
    B.height = HEIGHT;
    C.width = WIDTH;
    C.height = HEIGHT;

    // call to matrix multiplicaiton host method
    MatMul(A, B, C);

    // print matrices
    // for(i=0; i<HEIGHT ; ++i) {
    //     for(j=0; j<WIDTH; ++j) {
    //         printf("%f ", A.elements[i*HEIGHT + j]);
    //     }
    //     printf("\n");
    // }

    // printf("\n");
    
    // for(i=0; i<HEIGHT ; ++i) {
    //     for(j=0; j<WIDTH; ++j) {
    //         printf("%f ", B.elements[i*HEIGHT + j]);
    //     }
    //     printf("\n");
    // }

    // printf("\n");
    
    // for(i=0; i<HEIGHT ; ++i) {
    //     for(j=0; j<WIDTH; ++j) {
    //         printf("%f ", C.elements[i*HEIGHT + j]);
    //     }
    //     printf("\n");
    // }

    return 0;
}