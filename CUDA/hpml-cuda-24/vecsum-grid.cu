// Functionality: Add two given vectors
// Kernel specification: block size=NxNx1, N threads per block 

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N	3

__global__ void add( int *a, int *b, int *c ) {
	int bidx = blockIdx.x;
	int bidy = blockIdx.y;
	int tid = threadIdx.x;	// handle the data at this index
	int index = bidx*blockDim.x*gridDim.y + bidy*blockDim.x + tid;
	if(index<N*N*N) {
		c[index] = a[index] + b[index];
	}
	printf("blockIDx: %d, blockIDy: %d, threadIDx: %d index: %d\n", bidx, bidy, tid, index);
}

int main( void ) {
	int a[N*N*N], b[N*N*N], c[N*N*N];
	int *dev_a, *dev_b, *dev_c;

	// allocate the memory on the cpu
	hipMalloc((void**)&dev_a, N*sizeof(int)*N*N);
	hipMalloc((void**)&dev_b, N*sizeof(int)*N*N);
	hipMalloc((void**)&dev_c, N*sizeof(int)*N*N);

	for (int k=0; k<N; ++k)
	{
		for (int j=0; j<N; ++j)
		{
			for(int i=0; i<N; i++) {
				a[k*N*N+j*N+i] = -i;
				b[k*N*N+j*N+i] = i*i;
			}
		}
	}

	hipMemcpy(dev_a, a, N*sizeof(int)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N*sizeof(int)*N*N, hipMemcpyHostToDevice);

	dim3 gridDIM(N, N, 1);

	add<<<gridDIM,N>>>(dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, N*sizeof(int)*N*N, hipMemcpyDeviceToHost);


	for(int i=0; i<N*N*N; i++){
		printf( "%d + %d = %d\n", a[i], b[i], c[i]);
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
