// Funtionality: add two numbers using the GPU
// Kernel specification: one block, one thread

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c) {
	*c = a + b;
}

int main (void){
	int c;
	int *dev_c;
	hipMalloc((void**)&dev_c, sizeof(int));
	

	add<<<1,1>>>(2, 7, dev_c);

	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("2 + 7 = %d\n", c);
	hipFree(dev_c);

	return 0;
}
