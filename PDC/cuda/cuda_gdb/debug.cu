
// Program do vector addition. Use this for cucda-gdb debugging demo
/*
    compile: nvcc -g -G -allow-unsupported-compiler debug.cu -o debug
    run. cuda-gdb ./debug
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

#define ARRAY_SIZE 10000
#define THREADS_PER_BLOCK 128

__global__ void addArrays(int *a, int *b, int *c, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // condition omits the extra threads launched
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int a[ARRAY_SIZE], b[ARRAY_SIZE], c[ARRAY_SIZE];
    int *dev_a, *dev_b, *dev_c;

    // Initialize arrays a and b
    for (int i = 0; i < ARRAY_SIZE; ++i) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Allocate device memory
    hipMalloc((void**)&dev_a, ARRAY_SIZE * sizeof(int));
    hipMalloc((void**)&dev_b, ARRAY_SIZE * sizeof(int));
    hipMalloc((void**)&dev_c, ARRAY_SIZE * sizeof(int));

    // Copy data to device
    hipMemcpy(dev_a, a, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);


    // Calculate the number of blocks needed
    // THREADS_PER_BLOCK - 1 is usefull when ARRAY_SIZE is not a multiple of THREADS_PER_BLOCK. It will launch enough threads
    dim3 dimGrid((ARRAY_SIZE + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, 1, 1);
    dim3 dimBlock(THREADS_PER_BLOCK, 1, 1);
    // Launch kernel
    addArrays<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, ARRAY_SIZE);

    // Copy result back to host
    hipMemcpy(c, dev_c, ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    // Display the result
    for (int i = 0; i < ARRAY_SIZE; ++i) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
