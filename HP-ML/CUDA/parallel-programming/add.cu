
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add( int a, int b, int *c ) {
	*c = a + b;
}

int main ( void ) {
	int c;
	int *dev_c;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);


	hipMalloc((void**)&dev_c, sizeof(int));
	
	hipEventRecord(start);
	add<<<1,1>>>( 2, 7, dev_c );
	hipEventRecord(stop);

	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	printf( "2 + 7 = %d\n", c );
	printf("\n\nGPU running time: %f\n",milliseconds);

	hipFree( dev_c );

	return 0;
}
