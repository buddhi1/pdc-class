
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N	3

__global__ void add( int *a, int *b, int *c ) {
	int bidx = blockIdx.x;
	int bidy = blockIdx.y;
	int tid = threadIdx.x;	// handle the data at this index
	if(tid < N && bidx < N && bidy < N) {
		c[bidx*N*N + bidy*N + tid] = a[bidx*N*N + bidy*N + tid] + b[bidx*N*N + bidy*N + tid];
	}
	printf("blockIDx: %d, blockIDy: %d, threadIDx: %d\n", bidx, bidy, tid);
}

int main( void ) {
	int a[N*N*N], b[N*N*N], c[N*N*N];
	int *dev_a, *dev_b, *dev_c;

	// allocate the memory on the cpu
	hipMalloc((void**)&dev_a, N * sizeof(int) * N * N);
	hipMalloc((void**)&dev_b, N * sizeof(int) * N * N);
	hipMalloc((void**)&dev_c, N * sizeof(int) * N * N);

	for (int k = 0; k < N; ++k)
	{
		for (int j = 0; j < N; ++j)
		{
			for( int i = 0; i < N; i++ ) {
				a[k*N*N + j*N + i] = -i;
				b[k*N*N + j*N + i] = i * i;
			}
		}
	}

	hipMemcpy(dev_a, a, N * sizeof(int) * N * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int) * N * N, hipMemcpyHostToDevice);

	dim3 blockDIM(N, N, 1);

	add<<<blockDIM,N>>>( dev_a, dev_b, dev_c );

	hipMemcpy(c, dev_c, N * sizeof(int) * N * N, hipMemcpyDeviceToHost);


	for( int i = 0; i < N*N*N; i++ ){
		printf( "%d + %d = %d\n", a[i], b[i], c[i] );
	}

	hipFree( dev_a );
	hipFree( dev_b );
	hipFree( dev_c );

	return 0;
}
